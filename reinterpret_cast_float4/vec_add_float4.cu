
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

constexpr int N = 1 << 20;
constexpr int BlockSize = 256;
constexpr int NumIterations = 100;
constexpr int FloatPerThread = 4;
constexpr int GridSize = ((N + FloatPerThread - 1) / FloatPerThread + BlockSize - 1) / BlockSize;

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
} while(0)

__global__
void vec_add_float4(const float* a_ptr, const float* b_ptr, float* c_ptr, int n) {
    float a[FloatPerThread], b[FloatPerThread], c[FloatPerThread];

    const int idx = FloatPerThread * (blockDim.x * blockIdx.x + threadIdx.x);

    if (idx < n) {
        if (idx + FloatPerThread - 1 < n) {
            *reinterpret_cast<float4*>(a) = *reinterpret_cast<const float4*>(a_ptr + idx);
            *reinterpret_cast<float4*>(b) = *reinterpret_cast<const float4*>(b_ptr + idx);
        } else {
            for (int i = 0; i < FloatPerThread; ++i) {
                if (idx + i < n) {
                    a[i] = a_ptr[idx + i];
                    b[i] = b_ptr[idx + i];
                } else {
                    a[i] = 0.0f;
                    b[i] = 0.0f;
                }
            }
        }

        for (int i = 0; i < FloatPerThread; ++i) {
            c[i] = a[i] + b[i];
        }

        if (idx + FloatPerThread - 1 < n) {
            *reinterpret_cast<float4*>(c_ptr + idx) = *reinterpret_cast<const float4*>(c);
        } else {
            for (int i = 0; i < FloatPerThread; ++i) {
                if (idx + i < n) {
                    c_ptr[idx + i] = c[i];
                }
            }
        }
    }
}

int main() {
    size_t size = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void**)&d_A, size));
    CHECK_CUDA(hipMalloc((void**)&d_B, size));
    CHECK_CUDA(hipMalloc((void**)&d_C, size));
    CHECK_CUDA(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    std::cout << "=== float4 version ===" << std::endl;
    std::cout << "GridSize: " << GridSize << ", BlockSize: " << BlockSize << ", N: " << N << std::endl;
    std::cout << "Running " << NumIterations << " iterations for benchmarking..." << std::endl;
    
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    // Warmup run
    vec_add_float4<<<GridSize, BlockSize>>>(d_A, d_B, d_C, N);
    CHECK_CUDA(hipDeviceSynchronize());
    
    CHECK_CUDA(hipEventRecord(start));
    for (int iter = 0; iter < NumIterations; ++iter) {
        vec_add_float4<<<GridSize, BlockSize>>>(d_A, d_B, d_C, N);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    
    float total_milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&total_milliseconds, start, stop));
    
    float avg_milliseconds = total_milliseconds / NumIterations;
    
    std::cout << "\n=== Performance Results ===" << std::endl;
    std::cout << "Total time for " << NumIterations << " iterations: " << total_milliseconds << " ms" << std::endl;
    std::cout << "Average kernel execution time: " << avg_milliseconds << " ms" << std::endl;
    std::cout << "Effective bandwidth: " << (3 * N * sizeof(float)) / (avg_milliseconds * 1e6) << " GB/s" << std::endl;

    CHECK_CUDA(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    for (int i = 0; i < N; i++) {
        assert(h_C[i] == h_A[i] + h_B[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}