
#include <hip/hip_runtime.h>
#include <cassert>
#include <iostream>

constexpr int N = 1 << 20;
constexpr int BlockSize = 256;
constexpr int NumIterations = 100;
constexpr int FloatsPerThread = 4;
constexpr int GridSize = ((N + FloatsPerThread - 1) / FloatsPerThread + BlockSize - 1) / BlockSize;

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
} while(0)

__global__
void vec_add_by_4(const float* a_ptr, const float* b_ptr, float* c_ptr, int n) {
    float a[FloatsPerThread], b[FloatsPerThread], c[FloatsPerThread];

    const int idx = FloatsPerThread * (blockDim.x * blockIdx.x + threadIdx.x);

    if (idx < n) {
        if (idx + FloatsPerThread - 1 < n) {
            for (int i = 0; i < FloatsPerThread; ++i) {
                a[i] = a_ptr[idx + i];
            }

            for (int i = 0; i < FloatsPerThread; ++i) {
                b[i] = b_ptr[idx + i];
            }
        } else {
            for (int i = 0; i < FloatsPerThread; ++i) {
                if (idx + i < n) {
                    a[i] = a_ptr[idx + i];
                    b[i] = b_ptr[idx + i];
                } else {
                    a[i] = 0.0f;
                    b[i] = 0.0f;
                }
            }
        }

        for (int i = 0; i < FloatsPerThread; ++i) {
            c[i] = a[i] + b[i];
        }

        if (idx + FloatsPerThread - 1 < n) {
            for (int i = 0; i < FloatsPerThread; ++i) {
                c_ptr[idx + i] = c[i];
            }
        } else {
            for (int i = 0; i < FloatsPerThread; ++i) {
                if (idx + i < n) {
                    c_ptr[idx + i] = c[i];
                }
            }
        }
    }
}

int main() {
    size_t size = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc((void**)&d_A, size));
    CHECK_CUDA(hipMalloc((void**)&d_B, size));
    CHECK_CUDA(hipMalloc((void**)&d_C, size));
    CHECK_CUDA(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));

    std::cout << "=== vec_add_by_4 version (4 elements per thread) ===" << std::endl;
    std::cout << "GridSize: " << GridSize << ", BlockSize: " << BlockSize << ", N: " << N << std::endl;
    std::cout << "Running " << NumIterations << " iterations for benchmarking..." << std::endl;
    
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    // Warmup run
    vec_add_by_4<<<GridSize, BlockSize>>>(d_A, d_B, d_C, N);
    CHECK_CUDA(hipDeviceSynchronize());
    
    CHECK_CUDA(hipEventRecord(start));
    for (int iter = 0; iter < NumIterations; ++iter) {
        vec_add_by_4<<<GridSize, BlockSize>>>(d_A, d_B, d_C, N);
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));
    
    float total_milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&total_milliseconds, start, stop));
    
    float avg_milliseconds = total_milliseconds / NumIterations;
    
    std::cout << "\n=== Performance Results ===" << std::endl;
    std::cout << "Total time for " << NumIterations << " iterations: " << total_milliseconds << " ms" << std::endl;
    std::cout << "Average kernel execution time: " << avg_milliseconds << " ms" << std::endl;
    std::cout << "Effective bandwidth: " << (3 * N * sizeof(float)) / (avg_milliseconds * 1e6) << " GB/s" << std::endl;

    CHECK_CUDA(hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost));
    
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));

    for (int i = 0; i < N; i++) {
        assert(h_C[i] == h_A[i] + h_B[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}